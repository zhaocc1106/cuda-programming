#include "hip/hip_runtime.h"
/**
 * Cuda hello world，实现一个矩阵加法，并比较和cpu实现的耗时，发现矩阵较小时cpu可能快些，但较大后gpu快很多。
 */

#include <iostream>

#include "cuda_start.h"

// 检查两个float数组是否相同
void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8; // 错误容忍度
  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      printf("Results don\'t match!\n");
      printf("%f(hostRef[%d] )!= %f(gpuRef[%d])\n", hostRef[i], i, gpuRef[i], i);
      return;
    }
  }
  printf("Check result success!\n");
}

// 初始化数据
void initialData(float *ip, int size) {
  time_t t;
  srand((unsigned) time(&t));
  for (int i = 0; i < size; i++) {
    ip[i] = (float) (rand() & 0xffff) / 1000.0f;
  }
}

//CPU对照组，用于对比加速比
void sumMatrix2DonCPU(float *MatA, float *MatB, float *MatC, int nx, int ny) {
  float *a = MatA;
  float *b = MatB;
  float *c = MatC;
  for (int j = 0; j < ny; j++) {
    for (int i = 0; i < nx; i++) {
      c[i] = a[i] + b[i];
    }
    c += nx;
    b += nx;
    a += nx;
  }
}

//核函数，每一个线程计算矩阵中的一个元素。
__global__ void sumMatrix(const float *MatA, const float *MatB, float *MatC, int nx, int ny) {
  int ix = threadIdx.x + blockDim.x * blockIdx.x;
  int iy = threadIdx.y + blockDim.y * blockIdx.y;
  int idx = ix + iy * ny;
  if (ix < nx && iy < ny) {
    MatC[idx] = MatA[idx] + MatB[idx];
  }
}

//主函数
int main(int argc, char **argv) {
  //设备初始化
  printf("starting...\n");
  initDevice(0);

  //输入二维矩阵，4096*4096，单精度浮点型。
  int nx = 1 << 12;
  int ny = 1 << 12;
  int nBytes = nx * ny * sizeof(float);

  //Malloc，开辟主机内存
  float *A_host = (float *) malloc(nBytes);
  float *B_host = (float *) malloc(nBytes);
  float *C_host = (float *) malloc(nBytes);
  float *C_from_gpu = (float *) malloc(nBytes);
  initialData(A_host, nx * ny);
  initialData(B_host, nx * ny);

  //hipMalloc，开辟设备内存
  float *A_dev = nullptr;
  float *B_dev = nullptr;
  float *C_dev = nullptr;
  CHECK(hipMalloc((void **) &A_dev, nBytes));
  CHECK(hipMalloc((void **) &B_dev, nBytes));
  CHECK(hipMalloc((void **) &C_dev, nBytes));

  //输入数据从主机内存拷贝到设备内存
  CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

  //二维线程块，32×32
  dim3 block(32, 32);
  //二维线程网格，128×128
  dim3 grid((nx - 1) / block.x + 1, (ny - 1) / block.y + 1);

  //测试GPU执行时间
  double gpuStart = cpuSecond();
  //将核函数放在线程网格中执行
  sumMatrix<<<grid, block>>>(A_dev, B_dev, C_dev, nx, ny);

  CHECK(hipDeviceSynchronize());
  double gpuTime = cpuSecond() - gpuStart;
  printf("GPU Execution Time: %f sec\n", gpuTime);

  //在CPU上完成相同的任务
  hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost);
  double cpuStart = cpuSecond();
  sumMatrix2DonCPU(A_host, B_host, C_host, nx, ny);
  double cpuTime = cpuSecond() - cpuStart;
  printf("CPU Execution Time: %f sec\n", cpuTime);

  //检查GPU与CPU计算结果是否相同
  CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
  checkResult(C_host, C_from_gpu, nx * ny);

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_host);
  free(C_from_gpu);
  hipDeviceReset();
  return 0;
}